
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>

#define M 32
#define min(a,b) (((a)<(b))?(a):(b))

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return double(tv.tv_sec)+double(tv.tv_usec)*1e-6;
}

__global__ void matmul(float *A, float *B, float *C, int N) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  if(j < N) {
    __shared__ float s_A[M];
    for (int ks=0; ks<N+M-1; ks+=M) {
      __syncthreads();
      if (ks+threadIdx.x<N) s_A[threadIdx.x] = A[N*i+ks+threadIdx.x];
      else s_A[threadIdx.x] = 0.0;
      __syncthreads();
      for (int k=ks; k<min(ks+M,N); k++) {
        if(i==0&&j==32) printf("%d %f %f\n",k,A[N*i+k],s_A[k-ks]);
        sum += s_A[k-ks] * B[N*k+j];
      }
    }
  }
  C[N*i+j] = sum;
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float * h_D = new float [N*N];
  float *d_A, *d_B, *d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
      h_D[N*i+j] = 0;
    }
  }
  double tic = get_time();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  dim3 grid((N+M-1)/M, N);
  matmul<<<grid,M>>>(d_A, d_B, d_C, N);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  double toc = get_time();
  printf("N=%d: %lf s (%lf GFlops)\n",N,toc-tic,2.*N*N*N/(toc-tic)/1e9);
  tic = get_time();
#pragma omp parallel for
  for (int i=0; i<N; i++) {
    for (int k=0; k<N; k++) {
      for (int j=0; j<N; j++) {
        h_D[N*i+j] += h_A[N*i+k] * h_B[N*k+j];
      }
    }
  }
  toc = get_time();
  printf("N=%d: %lf s (%lf GFlops)\n",N,toc-tic,2.*N*N*N/(toc-tic)/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]-h_D[N*i+j]);
    }
  }
  printf("error: %f\n",err/N/N);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
}